#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <sstream>

//For memory leaks
#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>

//Sieves
#include "src/sieves/sieve_eratosthenes_cpu.h"

//Misc
inline void WaitForEnter() {
	std::string str;
	std::cout << "Enter to continue..." << std::endl;
	std::getline(std::cin, str);
}

int main() {
	//Check for memory leaks at each exit point of the program
	_CrtSetDbgFlag(_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF);
	//---
	std::cout << "<Program Start>" << std::endl;

	std::cout << SIZE_MAX << std::endl;
	std::cout << UINT_MAX << std::endl;

	SieveErathosthenesCPU eratosthenesA(10);
	std::cout << eratosthenesA.StringifyResults("ERATOSTHENES CPU") << std::endl;
	//std::cout << EratosthenesA.StringifyTrackerArr() << std::endl;

	//std::cout << ": " << eratosthenesA.IsPrime(2567) << std::endl;

	//---
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	std::cout << "<Program End>" << std::endl;

	WaitForEnter();

    return 0;
}



