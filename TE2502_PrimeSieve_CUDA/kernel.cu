#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <sstream>

//For memory leaks
#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>

//Sieves
#include "sieves/sieve_eratosthenes_cpu.h"

//Misc
inline void WaitForEnter() {
	std::string str;
	std::cout << "Enter to continue..." << std::endl;
	std::getline(std::cin, str);
}

int main() {
	//Check for memory leaks at each exit point of the program
	_CrtSetDbgFlag(_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF);
	//---

	SieveErathosthenesCPU EratosthenesA(23);

	std::cout << EratosthenesA.StringifyResults("ERATOSTHENES CPU") << std::endl;

	//---
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	WaitForEnter();

    return 0;
}
