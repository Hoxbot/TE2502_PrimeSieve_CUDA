#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <sstream>

//For memory leaks
#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>

//CPU Sieves
#include "src/sieves/sieve_eratosthenes_cpu.h"
#include "src/sieves/sieve_atkin_cpu.h"

//GPGPU Sieves
#include "src/sieves/sieve_sundaram_cuda.cuh"
#include "src/sieves/sieve_atkin_cuda.cuh"

//Misc
inline void WaitForEnter() {
	std::string str;
	std::cout << "Enter to continue..." << std::endl;
	std::getline(std::cin, str);
}

int main() {
	//Check for memory leaks at each exit point of the program
	_CrtSetDbgFlag(_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF);
	//---
	std::cout << "<Program Start>" << std::endl;

	/*
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	std::cout 
		<< "\t---CPU SIDE---\n"
		<< "\tMax allocation capacity (bytes):\t" << SIZE_MAX << "\n"
		<< "\t---CUDA SIDE---\n"
		<< "\tGlobal memory capacity (bytes):\t\t" << prop.totalGlobalMem << "\n"
		<< "\tShared memory capacity (bytes):\t\t" << prop.sharedMemPerBlock << "\n"
		<< "\tMax threads per block:\t\t\t" << prop.maxThreadsPerBlock << "\n";
	*/

	size_t n = 1024;

	//SieveErathosthenesCPU eratosthenesA(n);
	//std::cout << eratosthenesA.StringifyResults("ERATOSTHENES CPU") << std::endl;
	//std::cout << eratosthenesA.StringifyTrackerArr() << std::endl;

	//SieveSundaramCUDA sundaramA(n);
	//std::cout << sundaramA.StringifyResults("SUNDARAM GPGPU") << std::endl;
	//std::cout << sundaramA.StringifyTrackerArr() << std::endl;

	//SieveAtkinCUDA atkinA(n);
	//std::cout << atkinA.StringifyResults("ATKIN GPGPU") << std::endl;
	//std::cout << atkinA.StringifyTrackerArr() << std::endl;

	//----
	for (size_t i = 900; i < 2048; i++) {
		std::cout << "#########################################\n";
		std::cout << "CPU\n";
		std::cout << SieveAtkinCPU(i).StringifyResults(" ") << std::endl;
		std::cout << "CUDA\n";
		std::cout << SieveAtkinCUDA(i).StringifyResults(" ") << std::endl;
	}
	//----

	//Allocation test
	//std::cout << SieveSundaramCUDA(1024).StringifyResults("FIRST") << std::endl;
	//std::cout << SieveSundaramCUDA(1024).StringifyResults("SECOND") << std::endl;
	//std::cout << SieveSundaramCUDA(2048).StringifyResults("THIRD") << std::endl;
	//std::cout << SieveSundaramCUDA(4096).StringifyResults("FOURTH") << std::endl;

	//---
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	std::cout << "<Program End>" << std::endl;

	//WaitForEnter();

    return 0;
}



