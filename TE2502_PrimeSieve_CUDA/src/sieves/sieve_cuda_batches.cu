#include "hip/hip_runtime.h"
#include "sieve_cuda_batches.cuh"

#include "../support/cuda_error_output.h"

//Private------------------------------------------------------------------------------------------

//Protected----------------------------------------------------------------------------------------
void SieveCUDABatches::AllocateGPUMemory() {
	//Get GPU limitations
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	size_t gpu_global_mem_capacity = prop.totalGlobalMem;

	//Fetch the number of bytes stored on the CPU side memory
	size_t bytes_to_allocate = this->sieve_mem_ptr_->BytesAllocated();

	//If the more bytes are required than the GPU can hold
	//we index additional batches, partitioning the numbers
	size_t batch_num = 0;
	void* mem_ptr = this->sieve_mem_ptr_->getMemPtr();
	while (bytes_to_allocate > gpu_global_mem_capacity) {

		Batch b;

		size_t offset = batch_num * gpu_global_mem_capacity;

		b.batch_ptr = static_cast<bool*>(mem_ptr) + offset;	//Batch starts a number of bytes into cpu memory
		b.batch_size = gpu_global_mem_capacity;				//Size of batch
		b.batch_start_index = offset;						//The index (in cpu memory) the memory starts at

		this->batches_.push_back(b);

		batch_num++;
		bytes_to_allocate -= gpu_global_mem_capacity;
	}

	//Repeat process for the one batch that isn't overfull 
	Batch b;
	size_t offset = batch_num * gpu_global_mem_capacity;
	b.batch_ptr = static_cast<bool*>(mem_ptr) + offset;		//Batch starts a number of bytes into cpu memory
	b.batch_size = bytes_to_allocate;						//Size of batch
	b.batch_start_index = offset;							//The index (in cpu memory) the memory starts at
	this->batches_.push_back(b);
	batch_num++;

	//Allocate memory on device
	CUDAErrorOutput(
		hipMalloc(
		(void**)&(this->device_mem_ptr_),
			this->batches_[0].batch_size	//The first batch is always large enough. Either it is maxed out and
		),									//all other batches are the same or smaller, or it is the only batch.
		"hipMalloc()", __FUNCTION__
	);
}

void SieveCUDABatches::DeallocateGPUMemory() {
	//Deallocate the memory on device
	CUDAErrorOutput(
		hipFree(this->device_mem_ptr_),
		"hipFree()", __FUNCTION__
	);
	this->device_mem_ptr_ = nullptr;
}

void SieveCUDABatches::UploadMemory() {
	//Upload batch on given index
	CUDAErrorOutput(
		hipMemcpy(
			this->device_mem_ptr_,					//Target
			this->batches_[in_i].batch_ptr,			//Source
			this->batches_[in_i].batch_size,		//Byte count
			hipMemcpyHostToDevice					//Transfer type
		),
		"hipMemcpy()", __FUNCTION__
	);
}

void SieveCUDABatches::DownloadMemory() {
	//Download batch on given index
	CUDAErrorOutput(
		hipMemcpy(
			this->batches_[in_i].batch_ptr,			//Target
			this->device_mem_ptr_,					//Source
			this->batches_[in_i].batch_size,		//Byte count
			hipMemcpyHostToDevice					//Transfer type
		),
		"hipMemcpy()", __FUNCTION__
	);
}

void SieveCUDABatches::LaunchKernel(size_t in_sieve_start) {
	// Launch a kernel on the GPU with one thread for each element.
	//	->	block
	//	->	threads per block (max 1024)
	//	->	size of shared memory
		//NTS:	unsigned int, not size_t. Need to fix safe conversion?
		//		Excess threads are fine, cannot be more than 1024 which fits
	unsigned int full_blocks = this->sieve_mem_ptr_->NumberCapacity() / 1024;	//Number of full blocks
	unsigned int excess_threads = this->sieve_mem_ptr_->NumberCapacity() % 1024;		//Number of threads not handled by full blocks
	//size_t bytes = this->mem_class_ptr_->BytesAllocated();					//Number of bytes to be in shared memory //NTS: Everything is in global, no shared needed 

	//Get where sieving should end
	size_t n = this->sieve_mem_ptr_->NumberCapacity();

	//If there are to be several kernel launches we need to figure out
	//where the subsequent blocks should start
	size_t alt_start = in_sieve_start;

	//Launch full blocks with 1024 threads	
	//NTS: A kernel can have 48 blocks at maximum? : no : 2^31 - 1?
	unsigned int max_blocks = 2147483647;
	//size_t max_blocks = 2;
	while (full_blocks > 0) {

		//Determine number of blocks in launch
		unsigned int blocks_in_launch = (full_blocks > max_blocks) ? max_blocks : full_blocks;

		//Launch kernel
		std::cout << ">>\tLaunching [" << blocks_in_launch << " of " << full_blocks << "] full blocks\n";
		//SundaramKernel <<<blocks_in_launch, 1024, 0>>> (alt_start, n, this->device_mem_ptr_);
		this->SieveKernel(blocks_in_launch, 1024, alt_start, n, this->device_mem_ptr_);

		//Decrease number of remaining blocks
		//Move kernel starting value
		full_blocks -= blocks_in_launch;
		alt_start += blocks_in_launch * 1024;

		// Check for any errors launching the kernel
		CUDAErrorOutput(
			hipGetLastError(),
			"<full blocks launch>",
			__FUNCTION__
		);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		CUDAErrorOutput(
			hipDeviceSynchronize(),
			"hipDeviceSynchronize()",
			__FUNCTION__
		);

	}

	//Launch leftover threads in 1 block //NTS: Will run sequentially, thus start and end must be altered
	if (excess_threads > 0) {
		std::cout << ">>\tLaunching [" << excess_threads << "] excess threads\n";
		//SundaramKernel <<<1, excess_threads, 0>>> (alt_start, n, this->device_mem_ptr_);
		this->SieveKernel(1, excess_threads, alt_start, n, this->device_mem_ptr_);

		// Check for any errors launching the kernel
		CUDAErrorOutput(
			hipGetLastError(),
			"<excess thread launch>",
			__FUNCTION__
		);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		CUDAErrorOutput(
			hipDeviceSynchronize(),
			"hipDeviceSynchronize()",
			__FUNCTION__
		);
	}


}

//Public-------------------------------------------------------------------------------------------
SieveCUDABatches::SieveCUDABatches() {
}

SieveCUDABatches::~SieveCUDABatches() {
	//NTS: Do not delete this ptr here
	this->sieve_mem_ptr_ = nullptr;
}

void SieveCUDABatches::LinkMemory(PrimeMemoryBool * in_ptr) {
	this->sieve_mem_ptr_ = in_ptr;
}
