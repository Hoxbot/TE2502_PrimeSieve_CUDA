#include "hip/hip_runtime.h"
#include "sieve_sundaram_cuda_batches.cuh"

//CUDA---------------------------------------------------------------------------------------------
__global__ void SundaramBatchKernel(
	size_t in_start, 
	size_t in_end, 
	size_t in_generation, 
	size_t in_batch_size, 
	bool* in_device_memory
) {
	//Get the thread's index
	//unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;


	//Test: Flips every other generation true/false
	//in_device_memory[i] = (in_generation % 2 == 0);

	//---BATCH CURRENT GENERATION---
	//The first cuda thread has id 0
	//We offset by in_start (in the very beginning this is 1 since Sundaram starts at 1)
	i += in_start;

	//De-list all numbers that fullful the condition: (i + j + 2*i*j) <= n
	for (size_t j = i; (i + j + 2*i*j) <= in_end; j++) {
		in_device_memory[(i + j + 2*i*j) - in_start] = false;		// NTS: (-in_start) offsets to correct array index
	}
	
	//WORKING HERE: for several batches the catch up function does not do anything 

	//---BATCH EARLIER GENERATIONS---
	//>	Earlier batches would not have had access to the memory space of this batch.
	//>	For each earlier batch with the same thread position (ergo: i), find the
	//	first j that reaches into this batch's memory space
	//>	Iterate j:s until we reach the end of the batch
	for (size_t g = 0; g < in_generation; g++) {
		//Jump back one batch size to find the i of the previous generation
		i -= in_batch_size;

		//Compute which j is the first to reach into the current batch's memory space
		float j_start = ceilf((float)(in_start - i) / ((2 * i) + 1));
		//size_t a = in_start - i;
		//size_t b = 2 * i + 1;
		//float j_start = ceil((float)a/(float)b);
		
		//Remember that j >= i, so we never start from a j less than i
		j_start = fmaxf(j_start, i); 
		//If j_start is set to i
		//-> then 2*i + 2*i^2 > in_end
		//-> and the for-loop below breaks instantly

		//Run iterations until we reach the end of span (in_end)
		for (size_t j = j_start; (i + j + 2*i*j) <= in_end; j++) {
			in_device_memory[(i + j + 2*i*j) - in_start] = false;		// NTS: (-in_start) offsets to correct array index
		}
	}
	
}


//Private------------------------------------------------------------------------------------------
void SieveSundaramCUDABatches::SieveKernel(
	unsigned int in_blocks, 
	unsigned int in_threads, 
	size_t in_start, 
	size_t in_end, 
	size_t in_generation, 
	bool * in_mem_ptr
) {
	SundaramBatchKernel <<<in_blocks, in_threads, 0>>> (in_start, in_end, in_generation, this->threads_per_batch_, in_mem_ptr);
}

void SieveSundaramCUDABatches::DoSieve() {
	//Allocate
	this->AllocateGPUMemory(this->start_, this->end_);

	for (size_t i = 0; i < this->batches_.size(); i++) {
		//Upload batch
		this->UploadMemory(i);

		//Launch work-groups
		this->LaunchKernel(i);

		//Download batch
		this->DownloadMemory(i);
	}

	//Deallocate
	this->DeallocateGPUMemory();

}

size_t SieveSundaramCUDABatches::IndexToNumber(size_t in_i) {
	return 2 * (in_i + this->start_) + 1;
}

//Public-------------------------------------------------------------------------------------------
SieveSundaramCUDABatches::SieveSundaramCUDABatches(size_t in_n)// {
	: SieveBase(1, in_n), SieveCUDABatches() {

	//Determine memory capacity needed
	//NTS: +1 since we round up
	size_t mem_size = ((in_n - 2) / 2) + ((in_n - 2) % 2);

	this->mem_class_ptr_ = new PrimeMemoryBool(mem_size);
	//this->mem_class_ptr_ = new PrimeMemoryBit(mem_size);
	this->LinkMemory(this->mem_class_ptr_);

	//Sundaram starts all as primes
	this->mem_class_ptr_->SetAllPrime();

	this->timer_.SaveTime();

	this->DoSieve();

	this->timer_.SaveTime();
}

SieveSundaramCUDABatches::~SieveSundaramCUDABatches() {
	if (this->mem_class_ptr_ != nullptr) {
		delete this->mem_class_ptr_;
		this->mem_class_ptr_ = nullptr;
	}
}

bool SieveSundaramCUDABatches::IsPrime(size_t in_num) {
	//Everything outside scope is false
	if (in_num < this->start_ || in_num > this->end_) { return false; }

	//Sundaram's sieve does not store even numbers
	//> 2 special case
	//> All other even numbers false
	if (in_num == 2) { return true; }
	if ((in_num % 2) == 0) { return false; }

	//For odd numbers, offset number to correct index
	size_t the_number_index = ((in_num - 1) / 2) - this->start_;

	//Return
	return this->mem_class_ptr_->CheckIndex(the_number_index);

}
